#include "hip/hip_runtime.h"
#include <io.h>
#include <malloc.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void quickSort(int arr[], int left, int right);

int main(){
	int n=10;
	int index;
	int size;
	size = n * sizeof(int);
	int* h_arr = (int*)malloc(size); 
	
	int* d_arr;
	hipMalloc(&d_arr, size); 

	for(index=0;index<n;index++){
		printf("enter the %d number from arr",index);
		scanf("%d",&h_arr[index]);
	}

		
	hipMemcpy(d_arr, h_arr, size, hipMemcpyHostToDevice); 


	quickSort <<< 1, n >>>(d_arr, 0, n-1);
	 
	hipMemcpy(h_arr, d_arr, size, hipMemcpyDeviceToHost);
	hipFree(d_arr);

}


 __global__ void quickSort(int arr[], int left, int right) {

      int i = left, 
	  int j = right;
      int tmp;
	
      int pivot = arr[(left+right) / 2];
 

      while (i <= j) {
            while (arr[i] < pivot)
                  i++;
            while (arr[j] > pivot)
                  j--;
            if (i <= j) {
                  tmp = arr[i];
                  arr[i] = arr[j];
                  arr[j] = tmp;
                  i++;
                  j--;
            }
      };


      if (left < j)
            quickSort(arr, left, j);
      if (i < right)
            quickSort(arr, i, right);
}