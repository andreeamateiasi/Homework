#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cstdlib>
#include <ctime>

#define n 10

void random_matrix(int *array);
void print_matrix(int *array);
__global__ void filter(int *in, int *out);

int main(){
	int *in;
	int *out;
	int *dev_in, *dev_out;

	int size = n * n * sizeof(int);

	hipMalloc((void **)&dev_in, size);
	hipMalloc((void **)&dev_out, size);

	in = new int[size];
	out = new int[size];

	srand(time(nullptr));

	random_matrix(in);

	printf("\In:\n");
	print_matrix(in);

	hipMemcpy(dev_in, in, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_out, out, size, hipMemcpyHostToDevice);

	int numOfBlocks = 1;
	dim3 threadsPerBlock(n, n);

	filter << < numOfBlocks, threadsPerBlock >> > (dev_in, dev_out);

	hipMemcpy(out, dev_out, size, hipMemcpyDeviceToHost);

	printf("\Out: \n");
	print_matrix(out);

	hipFree(dev_in);
	hipFree(dev_out);

	free(in);
	free(out);
	int h;
	scanf("%d",&h);
	return 0;
}

void random_matrix(int *array){
	for (auto i = 0; i < n; ++i){
		for (auto j = 0; j < n; ++j){
			array[i * n + j] = rand() % 10;
		}
	}
}

void print_matrix(int *array){
	for (auto i = 0; i < n; ++i){
		for (auto j = 0; j < n; ++j){
			printf("%d ", array[i * n + j]);
		}

		printf("\n");
	}
}

__global__ void filter(int *in, int *out) {

	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	int start_i = 0, start_j = 0, sum = 0, end_i = n - 1, end_j = n - 1;

	if (i > 0) {
		start_i = i - 1;
	}
	if (j > 0) {
		start_j = j - 1;
	}
	if (i < n - 1) {
		end_i = i + 1;
	}
	if (j < n - 1) {
		end_j = j + 1;
	}

	int no_elements = 0;

	for (auto ir = start_i; ir <= end_i; ++ir) {
		for (auto ic = start_j; ic <= end_j; ++ic) {

			if (ic != j || ir != i) {
				sum += in[ir * n + ic];
				no_elements++;
			}
		}
	}

	out[i * n + j] = sum / no_elements;
}
