#include "hip/hip_runtime.h"

#ifndef __CUDACC__ 
#define __CUDACC__
#endif



#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include<time.h>
#include <stdio.h>
#include<malloc.h>

#include <stdio.h>
#include <time.h>
#define threads 10

#define SIZE 10
__global__ void find(int *a, int *elem, int *position){
    __shared__ int sdata[SIZE];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

	sdata[tid] = a[i];
	
	__syncthreads();
	for (unsigned int s = blockDim.x / 2; s >= 1; s = s / 2){
		if (tid < s){
			if (*elem == sdata[tid + s]){
				position[i] == tid + s;
				printf("pos for %d = %d\n",i, tid+s);
				i++;
			}
		}
		__syncthreads();
	}
	
}

int main(){
	int i, j, *h_elemToBeFound, *d_elemToBeFound;
	srand(time(NULL));

	int *host_a, *h_position;
	host_a = (int*)malloc(SIZE * sizeof(int));
	h_position = (int*)malloc(SIZE * sizeof(int));
	h_elemToBeFound = (int*)malloc(sizeof(int));
	int *dev_a;
	int *d_position;

	hipMalloc((void **)&dev_a, SIZE * sizeof(int));
	hipMalloc((void **)&d_position, SIZE * sizeof(int));

	hipMalloc((void **)&d_elemToBeFound, sizeof(int));

	*h_elemToBeFound = rand() % 10 + 1;

	printf("elem = %d", *h_elemToBeFound);
	for (i = 0; i < SIZE; i++){
		host_a[i] = rand() % 10 + 1;

	}
	for (i = 0; i < SIZE; i++){
		printf("%d ", host_a[i]);

	}
	/*for (i = 0; i < SIZE; i++)
		for (j = 0; j < SIZE; j++)
			h_position[i+j] == 0;*/
	printf(" ");
	hipMemcpy(d_position, h_position, SIZE * sizeof(int), hipMemcpyHostToDevice);

	hipMemcpy(dev_a, host_a, SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_elemToBeFound, h_elemToBeFound, sizeof(int), hipMemcpyHostToDevice);

	find<<<1, threads>>>(dev_a, d_elemToBeFound, d_position);

	hipMemcpy(h_position, d_position, SIZE * sizeof(int), hipMemcpyDeviceToHost);
	//cudaMemcpy(h_position, d_position, SIZE * sizeof(int), cudaMemcpyDeviceToHost);
	hipMemcpy(h_elemToBeFound, d_elemToBeFound, sizeof(int), hipMemcpyDeviceToHost);

	for (i = 0; i < SIZE; i++)
		//for (j = 0; j < SIZE; j++)
			printf("[%d] = %d",i,h_position[i ]);
	for (i = 0; i < SIZE; i++) {
		printf("%d ", host_a[i]);
	}

	hipFree(dev_a);
	hipFree(d_position);

	printf(" ");
	scanf("%d", i);
		return 0;
}