#include "hip/hip_runtime.h"
#include <io.h>
#include <malloc.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void quickSort(int arr[], int left, int right);

int main(){
	int n=10;
	int index;
	int size;
	size = n * sizeof(int);
	int* h_arr = (int*)malloc(size); 
	
	int* d_arr;
	hipMalloc(&d_arr, size); 

	for(index=0;index<n;index++){
		printf("enter the %d number from arr",index);
		scanf("%d",&h_arr[index]);
	}

		
	hipMemcpy(d_arr, h_arr, size, hipMemcpyHostToDevice); 


	quickSort <<< 1, n >>>(d_arr, 0, n-1);
	 
	hipMemcpy(h_arr, d_arr, size, hipMemcpyDeviceToHost);
	hipFree(d_arr);

}


 int partition (int *a,  int* b, int c, int left, int right){
	 int tmp,i,j,x;
	 x=a[right];
	 i=left - 1;
	 for( j = left; j<right; j++){
		 if(a[j]<x){
			 i++;
			 tmp = a[i];
			 a[i]=a[j];
			 a[j]=tmp;
		 }
	 }
	 i++;
   	 tmp = a[i];
	 a[i]=a[right];
	 a[right]=tmp;
	 return i;
 }
 __global__ void quickSort(int *a, int left, int right){
	 int q;
	 int  nleft, nright;
	 hipStream_t s1, s2;
	 q=partition( a+left, a+right, a[left], nleft, nright);

	 if(left<nright){
		 hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
		 quickSort<<<1, s1>>>(a, left, nright);
	 }
	 
	 if(nleft<right){
		 hipStreamCreateWithFlags(&s2, hipStreamNonBlocking);
		 quickSort<<<1, s2>>>(a, nleft, right);
	 }
 }



 /*
 int m(){
	 int a[10]={6,4,2,7,5,4,8,9,6,4},i;
	 for(i=0;i<=9;i++)
		 printf();
	 quickSort(a,0,9);
	 return 0;

 }*/





 /*
 __global__ void quickSort(int arr[], int left, int right) {

      int i = left, 
	  int j = right;
      int tmp;
	
      int pivot = arr[(left+right) / 2];
 

      while (i <= j) {
            while (arr[i] < pivot)
                  i++;
            while (arr[j] > pivot)
                  j--;
            if (i <= j) {
                  tmp = arr[i];
                  arr[i] = arr[j];
                  arr[j] = tmp;
                  i++;
                  j--;
            }
      };


      if (left < j)
            quickSort<<< 1, n >>>(arr, left, j);
      if (i < right)
            quickSort<<< 1, n >>>(arr, i, right);
}
*/






 void quick_p(char c[][30], int count)
{
   int partitions[1024];
   int newpartitions[1024];

   partitions[0]=0;
   partitions[1]=count-1;
   int threads=1;

   char *dev_c;
   hipMalloc((void**)&dev_c, count * sizeof( char));

   int *dev_p;
   int *dev_pn;
   hipMalloc((void**)&dev_p , 1024 * sizeof(int));
   hipMalloc((void**)&dev_pn, 1024 * sizeof(int));

   hipMemcpy(dev_c, c, count * sizeof( char), hipMemcpyHostToDevice);

   while(true)
   {
        hipMemcpy(dev_p, partitions , sizeof(partitions), hipMemcpyHostToDevice);
        hipMemcpy(dev_pn, newpartitions , sizeof(newpartitions), hipMemcpyHostToDevice);

        Split<<<1,threads>>>(&dev_c,dev_p,dev_pn,threads);

        // get result back and loop again
        hipMemcpy(newpartitions,dev_pn, sizeof(partitions), hipMemcpyDeviceToHost);

        int tmp=0;
        for(int i=0;i<threads*2;i++)
        {
            int idx=i*2;
            if (newpartitions[idx]<newpartitions[idx+1] && newpartitions[idx+1]-newpartitions[idx]>=1)
            {
                partitions[tmp]=newpartitions[idx];
                partitions[tmp+1]=newpartitions[idx+1];
                tmp+=2;
            }
        }
        threads=tmp/2;

        if (threads==0)
            break;
    } // end main loop

    hipMemcpy(c,dev_c, count* sizeof(unsigned char), hipMemcpyDeviceToHost);

    hipFree(dev_c);
    hipFree(dev_p);
    hipFree(dev_pn);
}