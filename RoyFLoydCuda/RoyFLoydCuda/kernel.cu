#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#ifndef __HIPCC__ 
#define __HIPCC__
#endif

#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

#include <stdio.h>
#define INF 999
#define n 5

__global__ void floyd(int *a) {
	int k;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	for (k = 0; k < n; k++)
		__syncthreads();
	if (a[i + k] + a[k + j] < a[i + j]) {
		a[i + j] = a[i + k] + a[k + j];
	}
}

void print(int **a) {
	int i, j;

	for (i = 0; i < n; i++)
		for (j = 0; j < n; j++)
			printf("%d", a[i*n + j]);
}
int main() {

	int  *d_a;
	int i, j, k;
	int size;
	size = n*n;

	int **h_a = (int**)malloc(n * sizeof(int));

	hipMalloc((void **)&d_a, size);

	for (i = 0; i < n; i++)
		for (j = 0; j < n; j++)
			h_a[i][j] = i + j;

	hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);

	dim3 threadBlock(n, n);


	floyd << <1, threadBlock >> >(d_a);

	hipMemcpy(h_a, d_a, size, hipMemcpyDeviceToHost);
	hipFree(d_a);
	print(h_a);

}
